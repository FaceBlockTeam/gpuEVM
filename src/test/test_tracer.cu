#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#ifndef __CGBN_H__
#define __CGBN_H__
#include <cgbn/cgbn.h>
#endif
#include "../stack.cuh"
#include "../tracer.cuh"
#include "../utils.h"

 
template<class params>
__global__ void kernel_tracer(cgbn_error_report_t *report,typename tracer_t<params>::tracer_content_t *tracers, typename stack_t<params>::stack_data_t *stacks, uint32_t instance_count) {
  typedef arith_env_t<params>                     arith_t;
  typedef typename arith_t::bn_t                  bn_t;
  typedef cgbn_mem_t<params::BITS>                evm_word_t;
  typedef stack_t<params>                         stack_t;
  typedef typename stack_t::stack_data_t          stack_data_t;
  typedef typename stack_t::stack_content_data_t  stack_content_data_t;
  typedef tracer_t<params>                        tracer_t;
  typedef typename tracer_t::tracer_content_t     tracer_content_t;
  typedef typename tracer_t::tracer_data_t        tracer_data_t;

  uint32_t instance=(blockIdx.x*blockDim.x + threadIdx.x)/params::TPI;
  __shared__ stack_content_data_t                 stack_content_data;
  
  if(instance>=instance_count)
    return;


  __syncthreads();
  if (threadIdx.x == 0)
    memcpy(&(stack_content_data.values[0]), stacks[instance].stack_base, sizeof(stack_content_data_t));
  __syncthreads();

  // setup arithmetic
  arith_t arith(cgbn_report_monitor, report, instance);
 
  //local_stack_t  stack(arith, &(instances[instance].values[0]), 0);
  stack_data_t   stack_data;
  stack_data.stack_offset = 0;
  stack_data.stack_base = &(stack_content_data.values[0]);

  stack_t  stack(arith, &(stack_data));
  tracer_t tracer(arith, &(tracers[instance]));

  uint32_t pc;
  pc=0;
  uint8_t opcode;
  opcode=OP_PUSH32;
  bn_t a, b, c, gas_cost, address;
  cgbn_set_ui32(arith._env, address, 1);
  cgbn_set_ui32(arith._env, gas_cost, 0);
  uint32_t error_code;
  cgbn_set_ui32(arith._env, a, instance);
  cgbn_set_ui32(arith._env, b, 0xFF);
  stack.push(b, error_code);
  tracer.push(address, pc, opcode, &stack);
  pc=1;
  stack.push(a, error_code);
  tracer.push(address, pc, opcode, &stack);
  pc=2;
  opcode=OP_SIGNEXTEND;
  stack.signextend(error_code);
  tracer.push(address, pc, opcode, &stack);
  stack.copy_stack_data(&(stacks[instance]), 0);
}

template<class params>
void run_test(uint32_t instance_count) {
  typedef arith_env_t<params>                     arith_t;
  typedef typename arith_t::bn_t                  bn_t;
  typedef cgbn_mem_t<params::BITS>                evm_word_t;
  typedef stack_t<params>                         stack_t;
  typedef typename stack_t::stack_data_t          stack_data_t;
  typedef typename stack_t::stack_content_data_t  stack_content_data_t;
  typedef tracer_t<params>                        tracer_t;
  typedef typename tracer_t::tracer_content_t     tracer_content_t;
  typedef typename tracer_t::tracer_data_t        tracer_data_t;
  
  stack_data_t          *cpu_stacks, *gpu_stacks;
  tracer_content_t      *cpu_tracers, *gpu_tracers;
  cgbn_error_report_t *report;
  arith_t arith(cgbn_report_monitor, 0);
  

  printf("Geenerating stack data ...\n");
  cpu_stacks=stack_t::get_stacks(instance_count);
  printf("Copying stack data to the GPU ...\n");
  gpu_stacks=stack_t::get_gpu_stacks(cpu_stacks, instance_count);
  printf("Freeing stack data on CPU ...\n");
  stack_t::free_stacks(cpu_stacks, instance_count);
  printf("Generating tracer data ...\n");
  cpu_tracers=tracer_t::get_tracers(instance_count);
  printf("Copying tracer data to the GPU ...\n");
  gpu_tracers=tracer_t::get_gpu_tracers(cpu_tracers, instance_count);
  printf("Freeing tracer data on CPU ...\n");
  tracer_t::free_tracers(cpu_tracers, instance_count);

  // create a cgbn_error_report for CGBN to report back errors
  CUDA_CHECK(cgbn_error_report_alloc(&report)); 
  
  printf("Running GPU kernel ...\n");

  kernel_tracer<params><<<instance_count, params::TPI>>>(report, gpu_tracers, gpu_stacks, instance_count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);
    
  // copy the instances back from gpuMemory
  printf("Copying results back to CPU ...\n");
  cpu_stacks=stack_t::get_cpu_stacks_from_gpu(gpu_stacks, instance_count);
  stack_t::free_gpu_stacks(gpu_stacks, instance_count);
  cpu_tracers=tracer_t::get_cpu_tracers_from_gpu(gpu_tracers, instance_count);
  
  // print the results
  printf("Printing results and create jsons\n");
  cJSON *root = cJSON_CreateObject();
  cJSON *post = cJSON_CreateArray();
  for(uint32_t instance=0; instance<instance_count; instance++) {
    stack_t local_stack(arith, &(cpu_stacks[instance]));
    tracer_t local_tracer(arith, &(cpu_tracers[instance]));
    printf("Instance %d:  ", instance);
    local_stack.print();
    printf("\n");
    local_tracer.print();
    printf("\n");
    cJSON *instance_json = cJSON_CreateObject();
    cJSON_AddItemToObject(instance_json, "stack", local_stack.to_json());
    cJSON_AddItemToObject(instance_json, "tracer", local_tracer.to_json());
    cJSON_AddItemToArray(post, instance_json);
  }
  printf("Results printed\n");
  cJSON_AddItemToObject(root, "post", post);
  char *json_str=cJSON_Print(root);
  FILE *fp=fopen("output/evm_tracer.json", "w");
  fprintf(fp, "%s", json_str);
  fclose(fp);
  free(json_str);
  cJSON_Delete(root);
  printf("Json files printed\n");
  
  
  // clean up
  printf("Freeing stack data on CPU ...\n");
  stack_t::free_stacks(cpu_stacks, instance_count);
  printf("Freeing tracer data on CPU ...\n");
  tracer_t::free_tracers(cpu_tracers, instance_count);
  printf("Freeing error report ...\n");
  CUDA_CHECK(cgbn_error_report_free(report));
}

int main() {
  run_test<utils_params>(2);
}