
#include "utils.h"
#include "evm.cuh"
#include <getopt.h>


template<class params>
void run_interpreter(char *read_json_filename, char *write_json_filename) {
  typedef evm_t<params> evm_t;
  typedef typename evm_t::evm_instances_t evm_instances_t;
  typedef arith_env_t<params> arith_t;
  
  evm_instances_t         cpu_instances;
  #ifndef ONLY_CPU
  evm_instances_t tmp_gpu_instances, *gpu_instances;
  #endif
  cgbn_error_report_t     *report;

  arith_t arith(cgbn_report_monitor, 0);
  
  //read the json file with the global state
  cJSON *read_root = get_json_from_file(read_json_filename);
  if(read_root == NULL) {
    printf("Error: could not read the json file\n");
    exit(EXIT_FAILURE);
  }
  cJSON *write_root = cJSON_CreateObject();
  const cJSON *test = NULL;
  cJSON_ArrayForEach(test, read_root) {
    // get instaces to run
    printf("Generating instances\n");
    evm_t::get_cpu_instances(cpu_instances, test);
    #ifndef ONLY_CPU
    CUDA_CHECK(hipDeviceReset());
    evm_t::get_gpu_instances(tmp_gpu_instances, cpu_instances);
    CUDA_CHECK(hipMalloc(&gpu_instances, sizeof(evm_instances_t)));
    CUDA_CHECK(hipMemcpy(gpu_instances, &tmp_gpu_instances, sizeof(evm_instances_t), hipMemcpyHostToDevice));
    #endif
    printf("Instances generated\n");

    // create a cgbn_error_report for CGBN to report back errors
    #ifndef ONLY_CPU
    CUDA_CHECK(cgbn_error_report_alloc(&report)); 
    size_t heap_size;
    hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
    CUDA_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, 1024*1024*1024));
    CUDA_CHECK(hipDeviceSetLimit(hipLimitStackSize, 64*1024));
    printf("Heap size: %zu\n", heap_size);
    hipDeviceGetLimit(&heap_size, hipLimitMallocHeapSize);
    printf("Heap size: %zu\n", heap_size);
    #endif

    #ifndef ONLY_CPU
    printf("Running GPU kernel ...\n");
    kernel_evm<params><<<cpu_instances.count, params::TPI>>>(report, gpu_instances);
    //CUDA_CHECK(hipPeekAtLastError());
    // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
    CUDA_CHECK(hipDeviceSynchronize());
    printf("GPU kernel finished\n");
    CGBN_CHECK(report);

    // copy the results back to the CPU
    printf("Copying results back to CPU\n");
    CUDA_CHECK(hipMemcpy(&tmp_gpu_instances, gpu_instances, sizeof(evm_instances_t), hipMemcpyDeviceToHost));
    evm_t::get_cpu_from_gpu_instances(cpu_instances, tmp_gpu_instances);
    printf("Results copied\n");
    #else
    printf("Running CPU EVM\n");
    // run the evm
    evm_t *evm = NULL;
    uint32_t tmp_error;
    for(uint32_t instance = 0; instance < cpu_instances.count; instance++) {
      printf("Running instance %d\n", instance);
      evm = new evm_t(
          arith,
          cpu_instances.world_state_data,
          cpu_instances.block_data,
          cpu_instances.sha3_parameters,
          &(cpu_instances.transactions_data[instance]),
          &(cpu_instances.accessed_states_data[instance]),
          &(cpu_instances.touch_states_data[instance]),
          #ifdef TRACER
          &(cpu_instances.tracers_data[instance]),
          #endif
          instance,
          &(cpu_instances.errors[instance]));
      evm->run(tmp_error);
      delete evm;
      evm = NULL;
    }
    printf("CPU EVM finished\n");
    #endif


    // print the results
    printf("Printing the results ...\n");
    evm_t::print_evm_instances_t(arith, cpu_instances);
    printf("Results printed\n");

    // print to json files
    printf("Printing to json files ...\n");
    cJSON_AddItemToObject(
      write_root,
      test->string,
      evm_t::json_from_evm_instances_t(arith, cpu_instances));
    printf("Json files printed\n");

    // free the memory
    printf("Freeing the memory ...\n");
    evm_t::free_instances(cpu_instances);
    #ifndef ONLY_CPU
    CUDA_CHECK(hipFree(gpu_instances));
    CUDA_CHECK(cgbn_error_report_free(report));
    CUDA_CHECK(hipDeviceReset());
    #endif
  }
  cJSON_Delete(read_root);
  char *json_str=cJSON_Print(write_root);
  FILE *fp=fopen(write_json_filename, "w");
  fprintf(fp, "%s", json_str);
  fclose(fp);
  free(json_str);
  cJSON_Delete(write_root);
}

int main(int argc, char *argv[]) {//getting the input
  char *read_json_filename = NULL;
  char *write_json_filename = NULL;
  static struct option long_options[] = {
        {"input", required_argument, 0, 'i'},
        {"output", required_argument, 0, 'o'},
        {0, 0, 0, 0}};

  int opt;
  int option_index = 0;
  while ((opt = getopt_long(argc, argv, "i:o:", long_options, &option_index)) != -1)
  {
      switch (opt)
      {
      case 'i':
          read_json_filename = optarg;
          break;
      case 'o':
          write_json_filename = optarg;
          break;
      default:
          fprintf(stderr, "Usage: %s --input <json_filename> --output <json_filename>\n", argv[0]);
          exit(EXIT_FAILURE);
      }
  }
  if (!read_json_filename || !write_json_filename)
  {
      fprintf(stderr, "Both --input and --output flags are required\n");
      exit(EXIT_FAILURE);
  }
  run_interpreter<utils_params>(read_json_filename, write_json_filename);
}